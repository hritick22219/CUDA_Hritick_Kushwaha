#include <stdlib.h>
#include <stdio.h>
#include <hipblas.h>
#define HA 2
#define WA 9
#define WB 2
#define HB WA 
#define WC WB   
#define HC HA  
#define index(i,j,ld) (((j)*(ld))+(i))

void printMat(float*P,int uWP,int uHP){
  int i,j;
  for(i=0;i<uHP;i++){

      printf("\n");

      for(j=0;j<uWP;j++)
          printf("%f ",P[index(i,j,uHP)]);
  }
}

__host__ float* initializeHostMemory(int height, int width, bool random, float nonRandomValue) {
  // TODO allocate host memory of type float of size height * width called hostMatrix

  // TODO fill hostMatrix with either random data (if random is true) else set each value to nonRandomValue

  return hostMatrix;
}

__host__ float *initializeDeviceMemoryFromHostMemory(int height, int width, float *hostMatrix) {
  // TODO allocate device memory of type float of size height * width called deviceMatrix

  // TODO set deviceMatrix to values from hostMatrix

  return deviceMatrix;
}

__host__ float *retrieveDeviceMemory(int height, int width, float *deviceMatrix, float *hostMemory) {
  // TODO get matrix values from deviceMatrix and place results in hostMemory

  return hostMemory;
}

__host__ void printMatrices(float *A, float *B, float *C){
  printf("\nMatrix A:\n");
  printMat(A,WA,HA);
  printf("\n");
  printf("\nMatrix B:\n");
  printMat(B,WB,HB);
  printf("\n");
  printf("\nMatrix C:\n");
  printMat(C,WC,HC);
  printf("\n");
}

__host__ int freeMatrices(float *A, float *B, float *C, float *AA, float *BB, float *CC){
  free( A );  free( B );  free ( C );
  hipblasStatus_t status = cublasFree(AA);
  if (status != HIPBLAS_STATUS_SUCCESS) {
    fprintf (stderr, "!!!! memory free error (A)\n");
    return EXIT_FAILURE;
  }
  status = cublasFree(BB);
  if (status != HIPBLAS_STATUS_SUCCESS) {
    fprintf (stderr, "!!!! memory free error (B)\n");
    return EXIT_FAILURE;
  }
  status = cublasFree(CC);
  if (status != HIPBLAS_STATUS_SUCCESS) {
    fprintf (stderr, "!!!! memory free error (C)\n");
    return EXIT_FAILURE;
  }
  return EXIT_SUCCESS;
}

int  main (int argc, char** argv) {
  hipblasStatus_t status;
  cublasInit();

  // TODO initialize matrices A and B (2d arrays) of floats of size based on the HA/WA and HB/WB to be filled with random data

  if( A == 0 || B == 0){
    return EXIT_FAILURE;
  } else {
    // TODO create arrays of floats C filled with random value
    // TODO create arrays of floats alpha filled with 1's
    // TODO create arrays of floats beta filled with 0's

    // TODO use initializeDeviceMemoryFromHostMemory to create AA from matrix A
    // TODO use initializeDeviceMemoryFromHostMemory to create BB from matrix B
    // TODO use initializeDeviceMemoryFromHostMemory to create CC from matrix C

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // TODO perform Single-Precision Matrix to Matrix Multiplication, GEMM, on AA and BB and place results in CC

    C = retrieveDeviceMemory(HC, WC, CC, C);

    printMatrices(A, B, C);

    freeMatrices(A, B, C, AA, BB, CC);
    
    /* Shutdown */
    status = cublasShutdown();
    if (status != HIPBLAS_STATUS_SUCCESS) {
      fprintf (stderr, "!!!! shutdown error (A)\n");
      return EXIT_FAILURE;
    }

    return EXIT_SUCCESS;
  }

}
